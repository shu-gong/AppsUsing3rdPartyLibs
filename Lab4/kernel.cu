#include "hip/hip_runtime.h"
﻿/*
Author: Shu Gong
Class: ECE6122 QSZ
Last Date Modified: 11/12/2022

Description:

a C\C++ program using CUDA to solve the steady state temperature distribution in the thin plate.

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<iostream>
#include ""
#include<fstream>
#include<string>
#include <iomanip>

#include <unistd.h>
#include <sys/types.h>
using namespace std;

/******************************************************************/
// @brief: set up dimGrid
// @para: a-> nInterior, b->dimBlock
/******************************************************************/
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

/******************************************************************/
// @brief: use CUDA to calculate temperature
// @para: g,h are device memory
/******************************************************************/
__global__ void temperatureCalc(double* g, double* h, int nInterior)
{
    int NX = nInterior + 2;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    //                         N 
    int P = i + j * NX;             // node (i,j)              |
    int N = i + (j + 1) * NX;       // node (i,j+1)            |
    int S = i + (j - 1) * NX;       // node (i,j-1)     W ---- P ---- E
    int E = (i + 1) + j * NX;       // node (i+1,j)            |
    int W = (i - 1) + j * NX;       // node (i-1,j)            |
    //                         S 

// --- Only update "interior" (not boundary) node points
    if (i > 0 && i < nInterior + 1 && j>0 && j < nInterior + 1)
        g[P] = 0.25 * (h[E] + h[W] + h[N] + h[S]);

}

int main(int argc, char* argv[])
{

    int nInterior;
    int nIteration;
    extern char* optarg;
    extern int optind, opterr, optopt;

    int c;
    const char* optstring = "n:I:";
    while ((c = getopt(argc, argv, optstring)) != -1)
    {
        switch (c)
        {
        case 'n':
            nInterior = atoi(optarg);
            if (nInterior <= 0 || !nInterior)
            {
                printf("Invalid parameters, please check your values.");
            }
            break;
        case 'I':
            nIteration = atoi(optarg);
            if (nIteration <= 0 || !nIteration)
            {
                printf("Invalid parameters, please check your values.");
            }
            break;
        case '?':
            printf("error optopt: %c\n", optopt);
            printf("error opterr: %d\n", opterr);
            break;
        }
    }


    int N = (nInterior + 2) * (nInterior + 2);

    double* g, * h, * out;
    double* d_g, * d_h;

    // Allocate memory
    g = (double*)calloc(N, sizeof(double));
    h = (double*)calloc(N, sizeof(double));
    out = (double*)malloc(sizeof(double) * N);

    // Initialize array
    for (int i = 0; i < N; i++)
    {
        if (i < 7 * (nInterior) / 10 && i>3 * (nInterior) / 10)
        {
            h[i] = 100.0;
            g[i] = 100.0;
        }
        else
        {
            h[i] = 20.0;
            g[i] = 20.0;
        }
    }

    // Allocate device memory 
    hipMalloc((void**)&d_g, sizeof(double) * N);
    hipMalloc((void**)&d_h, sizeof(double) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_g, g, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_h, d_g, sizeof(double) * N, hipMemcpyDeviceToDevice);

    // Executing kernel 
    dim3 dimBlock(15, 15);
    dim3 dimGrid(iDivUp((nInterior + 2), 15), iDivUp((nInterior + 2), 15));

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    for (int k = 0; k < nIteration; k = k + 2)
    {
        temperatureCalc << <dimGrid, dimBlock >> > (d_g, d_h, nInterior);
        temperatureCalc << <dimGrid, dimBlock >> > (d_h, d_g, nInterior);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);

    std::cout << "Thin plate calculation took " << time << " milliseconds." << std::endl;

    // Transfer data back to host memory
    hipMemcpy(out, d_g, sizeof(double) * N, hipMemcpyDeviceToHost);

    fstream outFile;
    outFile.open("finalTemperatures.csv", ios::out);
    for (int i = 0; i < nInterior + 2; i++)
    {
        for (int j = 0; j < nInterior + 2; j++)
        {
            outFile << std::setprecision(15) << out[j + i * (nInterior + 2)] << ",";
        }
        outFile << endl;
    }
    outFile.close();


    // Deallocate device memory
    hipFree(d_g);
    hipFree(out);

    // Deallocate host memory
    free(g);
    free(h);
}
